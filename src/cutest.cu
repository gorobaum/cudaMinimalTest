#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cuda_occupancy.h"

#include "cutest.h"

__global__ void testCuda(int* cudamtxa, int* cudamtxb, int* cudamtxc) {
  int i = threadIdx.x;

  cudamtxc[i] = cudamtxa[i] + cudamtxb[i];
}

void runCudaTest(int size) {
  int *a, *b, *c;
  int *cudamtxa, *cudamtxb, *cudamtxc;

  dim3 threadPerBlock(size);

  a = (int*)malloc(size * sizeof(int));
  b = (int*)malloc(size * sizeof(int));
  c = (int*)malloc(size * sizeof(int));

  for (int i = 0; i < size; i++) {
    a[i] = 1;
    b[i] = i;
  }

  hipMalloc(&cudamtxa, size*sizeof(int));
  hipMalloc(&cudamtxb, size*sizeof(int));
  hipMalloc(&cudamtxc, size*sizeof(int));

  hipMemcpy(cudamtxa, a, size*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(cudamtxb, b, size*sizeof(int), hipMemcpyHostToDevice);

  testCuda<<<1, threadPerBlock>>>(cudamtxa, cudamtxb, cudamtxc);
  hipMemcpy(c, cudamtxc, size*sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  for (int i = 0; i < size; i++)
    std::cout << c[i] << std::endl;

  free(a);
  free(b);
  free(c);
}
